#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "training.cuh"

static void updateNeuronErrorsLaunch(Layer* layer, const int& start_chunk, const int& end_chunk,
  const float* expected_output);
static void updateNeuronDeltasLaunch(Layer* layer, const int& start_chunk, const int& end_chunk,
  const float* input);
static float sumAllErrors(const int& start_chunk, const int& end_chunk, Neuron** neurons);

__global__ void updateNeuronErrors(const int neuron_num, const float* expected_output,
  const float* outputs, const float* gammas, float* errors, float* biases)
{
  float output;

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < neuron_num; i += stride)
  {
    output = outputs[i];

    errors[i] = output * (1 - output) * (expected_output[i] - output);
    biases[i] += gammas[i] * errors[i];
  }
}

__global__ void updateNeuronDeltas(const int input_num, float* weights, float* deltas,
  const float error, const float* input, const float alpha, const float gamma)
{
  float delta;

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < input_num; i += stride)
  {
    delta = deltas[i] * alpha + gamma * error * input[i];

    deltas[i] = delta;
    weights[i] += delta;
  }
}

void updateNeuronErrorsLaunch(Layer* layer, const int& start_chunk, const int& end_chunk,
  const float* expected_output)
{
  int gridSize;
  int minGridSize = 0;
  int blockSize = 0;
  int neuron_num = end_chunk - start_chunk;

  float* expected_output_d;
  float* outputs_d;
  float* errors_d;
  float* biases_d;
  float* gammas_d;

  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, updateNeuronErrors, 0, 0);
  gridSize = (neuron_num + blockSize - 1) / blockSize;

  hipMalloc(&expected_output_d, neuron_num * sizeof(float));
  hipMemcpy(expected_output_d, expected_output, neuron_num * sizeof(float), hipMemcpyHostToDevice);

  hipMalloc(&outputs_d, neuron_num * sizeof(float));
  hipMalloc(&errors_d, neuron_num * sizeof(float));
  hipMalloc(&biases_d, neuron_num * sizeof(float));
  hipMalloc(&gammas_d, neuron_num * sizeof(float));

  for (int i = 0; i < neuron_num; i++)
  {
    hipMemcpy(&(outputs_d[i]), &(layer->neurons_[start_chunk + i]->output_), sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&(gammas_d[i]), &(layer->neurons_[start_chunk + i]->gamma_), sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&(errors_d[i]), &(layer->neurons_[start_chunk + i]->error_), sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&(biases_d[i]), &(layer->neurons_[start_chunk + i]->bias_), sizeof(float), hipMemcpyHostToDevice);
  }

  updateNeuronErrors<<<gridSize, blockSize>>>(neuron_num, expected_output_d,
    outputs_d, gammas_d, errors_d, biases_d);

  for (int i = 0; i < neuron_num; i++)
  {
    hipMemcpy(&(layer->neurons_[start_chunk + i]->error_), &(errors_d[i]), sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&(layer->neurons_[start_chunk + i]->bias_), &(biases_d[i]), sizeof(float), hipMemcpyDeviceToHost);
  }

  hipFree(outputs_d);
  hipFree(gammas_d);
  hipFree(errors_d);
  hipFree(biases_d);
  hipFree(expected_output_d);
}

void updateNeuronDeltasLaunch(Layer* layer, const int& start_chunk, const int& end_chunk,
  const float* input)
{
  int gridSize;
  int minGridSize = 0;
  int blockSize = 0;
  int input_num = layer->neurons_[0]->input_num_;

  float* input_d;
  float* weights_d;
  float* deltas_d;

  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, updateNeuronDeltas, 0, 0);
  gridSize = (input_num + blockSize - 1) / blockSize;

  hipMalloc(&input_d, input_num * sizeof(float));
  hipMemcpy(input_d, input, input_num * sizeof(float), hipMemcpyHostToDevice);

  for (int i = start_chunk; i < end_chunk; i++)
  {
    hipMalloc(&weights_d, input_num * sizeof(float));
    hipMalloc(&deltas_d, input_num * sizeof(float));

    hipMemcpy(weights_d, layer->neurons_[i]->weights_, input_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deltas_d, layer->neurons_[i]->deltas_, input_num * sizeof(float), hipMemcpyHostToDevice);

    updateNeuronDeltas<<<gridSize, blockSize>>>(input_num, weights_d, deltas_d,
      layer->neurons_[i]->error_, input_d, layer->neurons_[i]->alpha_, layer->neurons_[i]->gamma_);

    hipMemcpy(layer->neurons_[i]->weights_, weights_d, input_num * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(layer->neurons_[i]->deltas_, deltas_d, input_num * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(weights_d);
    hipFree(deltas_d);
  }

  hipFree(input_d);
}

// TODO: Use OMP to sum
float sumAllErrors(const int& start_chunk, const int& end_chunk, Neuron** neurons)
{
  int input_num = neurons[0]->input_num_;
  float sum;

  for (int i = start_chunk; i < end_chunk; i++)
    for (int j = 0; j < input_num; j++)
      sum += neurons[i]->error_ * neurons[i]->weights_[j];

  return sum;
}

void parallelTraining(Layer* layer, const int& start_chunk, const int& end_chunk,
  const float* input, const float* expected_output)
{
  float error_sum;

  updateNeuronErrorsLaunch(layer, start_chunk, end_chunk, expected_output);
  updateNeuronDeltasLaunch(layer, start_chunk, end_chunk, input);

  hipStreamSynchronize(0);  // XXX: hipDeviceSynchronize() ?

  error_sum = sumAllErrors(start_chunk, end_chunk, layer->neurons_);

  {
    std::lock_guard<std::mutex> lock(layer->mutex_);
    layer->shared_error_sum_ += error_sum;
  }
}